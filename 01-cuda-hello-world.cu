
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vecAdd(double *a, double *b, double *c, size_t n)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main()
{
    size_t n = 50000000;
    size_t bytes = n * sizeof(double);
    double *h_a = (double *) malloc(bytes);
    double *h_b = (double *) malloc(bytes);
    double *h_c = (double *) malloc(bytes);  // output vector
    for (int i = 0; i < n; i++) {
        h_a[i] = sin(i) * sin(i);
        h_b[i] = cos(i) * cos(i);
    }

    double *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    const int blockSize = 1024;
    const int gridSize = (int)ceil((float)n/blockSize);

    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    double sum = 0;
    for (int i = 0; i < n; i++) {
        sum += h_c[i];
    }

    printf("final result: %f\n", sum / n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}

